#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/l2_norm_layer.hpp" 
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void L2NormLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* norm_scale = norm_.mutable_cpu_data();
  Dtype normsqr;
  int n = bottom[0]->num();
  int d = bottom[0]->count() / n;
  caffe_gpu_powx(n*d, bottom_data, Dtype(2), top_data);
  for (int i=0; i<n; ++i) {
    caffe_gpu_asum<Dtype>(d, top_data+i*d, &normsqr);
    if (normsqr < eps_) normsqr = eps_;
    norm_scale[i] = pow(normsqr, Dtype(-0.5));
    caffe_gpu_scale<Dtype>(d, norm_scale[i], bottom_data + i*d, top_data + i*d);
  }
}

template <typename Dtype>
void L2NormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* norm_scale = norm_.cpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int n = top[0]->num();
  const int d = top[0]->count() / n;
  Dtype a;
  caffe_copy(bottom[0]->count(), top_diff, bottom_diff);
  for (int i=0; i<n; ++i) {
    caffe_gpu_dot(d, top_data+i*d, top_diff+i*d, &a);
    caffe_gpu_axpby(d, Dtype(-1) * a * norm_scale[i], top_data + i*d, norm_scale[i], bottom_diff + i*d);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(L2NormLayer);
}  // namespace caffe
